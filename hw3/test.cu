#include <hip/hip_runtime.h>
#include <iostream>

// CUDA 核函数，计算向量加法
__global__ void vecAdd(float *A, float *B, float *C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1000;  // 向量大小
    size_t size = N * sizeof(float);

    // 在主机端分配内存
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // 初始化向量
    for (int i = 0; i < N; i++) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // 在设备端分配内存
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // 将数据从主机传输到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 设置 CUDA 网格和块维度
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // 启动向量加法核函数
    vecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // 将结果从设备传回主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 输出前 10 个结果
    for (int i = 0; i < 10; i++) {
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
    }

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // 释放主机内存
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
